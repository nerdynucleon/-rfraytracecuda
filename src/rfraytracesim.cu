#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void computeRays() {
    printf("Hello from my kernel\n");
}

int rfraytrace(){
    computeRays<<<1,1>>>();
    hipDeviceSynchronize();
    printf("Hello  from rfraytrace!\n");
    return 0;
}